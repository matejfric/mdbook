#include "hip/hip_runtime.h"
#include <glew.h>
#include <freeglut.h>
#include <cudaDefs.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h> // normalize method
#include <imageManager.h>
#include <benchmark.h>

#define TPB_1D 8              // ThreadsPerBlock in one dimension
#define TPB_2D TPB_1D *TPB_1D // ThreadsPerBlock = TPB_1D*TPB_1D (2D block)

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

using DT = uchar4;

// OpenGL
struct GLData
{
    unsigned int imageWidth;
    unsigned int imageHeight;
    unsigned int imageBPP; // Bits Per Pixel = 8, 16, 24, or 32 bit
    unsigned int imagePitch;

    unsigned int pboID;
    unsigned int textureID;
    unsigned int viewportWidth = 1024;
    unsigned int viewportHeight = 1024;
};
GLData gl;

unsigned char someValue = 0;

// CUDA
struct CudaData
{
    hipTextureDesc texDesc; // Texture descriptor used to describe texture parameters

    hipArray_t texArrayData;             // Source texture data
    hipResourceDesc resDesc;             // A resource descriptor for obtaining the texture data
    hipChannelFormatDesc texChannelDesc; // Texture channel descriptor to define channel bytes
    hipTextureObject_t texObj;           // Cuda Texture Object to be produces

    hipGraphicsResource_t texResource;
    hipGraphicsResource_t pboResource; // Buffer object

    CudaData()
    {
        memset(this, 0, sizeof(CudaData)); // DO NOT DELETE THIS !!!
    }
};

CudaData cd;

#pragma region CUDA Routines

__global__ void applyFilter(
    const hipTextureObject_t srcTex,
    const unsigned char someValue,
    const unsigned int pboWidth,
    const unsigned int pboHeight,
    unsigned char *pbo)
{
    // Make some data processing
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;

    if ((tx >= pboWidth) || (ty >= pboHeight))
        return;

    // Read from texture
    uchar4 texel = tex2D<uchar4>(srcTex, tx, ty);

    // Write to PBO
    int pboIdx = (ty * pboWidth + tx) * 4; // buffer is RGBA
    pbo[pboIdx + 0] = (someValue + pboIdx) % 255;
    pbo[pboIdx + 1] = texel.y;
    pbo[pboIdx + 2] = texel.z;
    pbo[pboIdx + 3] = texel.w;
}

void cudaWorker()
{
    // Map GL resources (TEXTURE and PBO)
    checkCudaErrors(hipGraphicsMapResources(
        1, &cd.texResource, 0));
    checkCudaErrors(hipGraphicsSubResourceGetMappedArray(
        &cd.texArrayData, cd.texResource, 0, 0));
    checkCudaErrors(hipGraphicsMapResources(
        1, &cd.pboResource, 0));

    uint8_t *pboData; // We wanna read byte by byte.
    size_t pboSizeBytes = 0;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer(
        (void **)&pboData, &pboSizeBytes, cd.pboResource));

    // Run kernel
    dim3 block(TPB_1D, TPB_1D);
    dim3 grid((gl.imageWidth + block.x - 1) / block.x, (gl.imageHeight + block.y - 1) / block.y);
    unsigned char someValue = 42;
    applyFilter<<<grid, block>>>(cd.texObj, someValue, gl.imageWidth, gl.imageHeight, pboData);

    // Unmap GL Resources (TEXTURE + PBO)
    checkCudaErrors(hipGraphicsUnmapResources(
        1, &cd.texResource, 0));
    checkCudaErrors(hipGraphicsUnmapResources(
        1, &cd.pboResource, 0));

    // This updates GL texture from PBO (copy PBO to texture)
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, gl.pboID);
    glBindTexture(GL_TEXTURE_2D, gl.textureID);
    glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, gl.imageWidth, gl.imageHeight, GL_RGBA, GL_UNSIGNED_BYTE, NULL); // Source parameter is NULL, Data is coming from a PBO, not host memory

    printf("."); // "progress bar"
}

void initCUDAObjects()
{
    // Register image to CUDA tex resource.
    hipGraphicsGLRegisterImage(&cd.texResource, gl.textureID, GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly);

    // Map resource and retrieve pointer to underlying array data.
    hipGraphicsMapResources(1, &cd.texResource, 0);
    hipGraphicsSubResourceGetMappedArray(&cd.texArrayData, cd.texResource, 0, 0);

    // Set resource descriptor.
    cd.resDesc.resType = hipResourceTypeArray;
    cd.resDesc.res.array.array = cd.texArrayData;

    // Set Texture Descriptor: Tex Units will know how to read the texture.
    // Set the readMode, normalizedCoords, filterMode, addressMode for each dimension.
    cd.texDesc.addressMode[0] = hipAddressModeClamp;
    cd.texDesc.addressMode[1] = hipAddressModeClamp;
    cd.texDesc.filterMode = hipFilterModePoint;
    cd.texDesc.readMode = hipReadModeElementType;
    cd.texDesc.normalizedCoords = 0; // false

    // Set Channel Descriptor: How to interpret individual bytes.
    // Retrieve the data from cd.texArrayData.
    checkCudaErrors(hipGetChannelDesc(
        &cd.texChannelDesc, cd.texArrayData));

    // Create CUDA Texture Object.
    checkCudaErrors(hipCreateTextureObject(
        &cd.texObj, &cd.resDesc, &cd.texDesc, NULL));

    // Unmap resource: Release the resource for OpenGL.
    checkCudaErrors(hipGraphicsUnmapResources(
        1, &cd.texResource, 0));

    // Register PBO.
    checkCudaErrors(hipGraphicsGLRegisterBuffer(
        &cd.pboResource, gl.pboID, cudaGraphicsMapFlagsWriteDiscard));
}

void releaseCUDA()
{
    // Unregister resources
    hipGraphicsUnregisterResource(cd.texResource);
    hipGraphicsUnregisterResource(cd.pboResource);
}
#pragma endregion

#pragma region OpenGL Routines
void prepareGlObjects(const char *imageFileName)
{
    FIBITMAP *tmp = ImageManager::GenericLoader(imageFileName, 0);
    gl.imageWidth = FreeImage_GetWidth(tmp);
    gl.imageHeight = FreeImage_GetHeight(tmp);
    gl.imageBPP = FreeImage_GetBPP(tmp);
    gl.imagePitch = FreeImage_GetPitch(tmp);

    // OpenGL Texture
    glEnable(GL_TEXTURE_2D);
    glGenTextures(1, &gl.textureID);
    glBindTexture(GL_TEXTURE_2D, gl.textureID);

    // WARNING: Just some of inner format are supported by CUDA!!!
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, gl.imageWidth, gl.imageHeight, 0, GL_BGRA, GL_UNSIGNED_BYTE, FreeImage_GetBits(tmp));
    // How to access the texture - linear and clamp
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);

    FreeImage_Unload(tmp);

    glGenBuffers(1, &gl.pboID);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, gl.pboID);                                                  // Make this the current UNPACK buffer (OpenGL is state-based)
    glBufferData(GL_PIXEL_UNPACK_BUFFER, gl.imageWidth * gl.imageHeight * 4, NULL, GL_DYNAMIC_COPY); // Allocate data for the buffer. 4-channel 8-bit image
}

void my_display()
{
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    glEnable(GL_TEXTURE_2D);
    glBindTexture(GL_TEXTURE_2D, gl.textureID);

    glBegin(GL_QUADS);

    // Texture coordinates and viewport coordinates
    glTexCoord2d(0, 0);
    glVertex2d(0, 0);
    glTexCoord2d(1, 0);
    glVertex2d(gl.viewportWidth, 0);
    glTexCoord2d(1, 1);
    glVertex2d(gl.viewportWidth, gl.viewportHeight);
    glTexCoord2d(0, 1);
    glVertex2d(0, gl.viewportHeight);

    glEnd();

    glDisable(GL_TEXTURE_2D);

    glFlush();
    glutSwapBuffers();
}

void my_resize(GLsizei w, GLsizei h)
{
    gl.viewportWidth = w;
    gl.viewportHeight = h;

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    glViewport(0, 0, gl.viewportWidth, gl.viewportHeight);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluOrtho2D(0, gl.viewportWidth, 0, gl.viewportHeight);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glutPostRedisplay();
}

void my_idle()
{
    cudaWorker();
    glutPostRedisplay();
}

void initGL(int argc, char **argv)
{
    glutInit(&argc, argv);

    glutInitDisplayMode(GLUT_RGBA | GLUT_DEPTH | GLUT_DOUBLE);
    glutInitWindowSize(gl.viewportWidth, gl.viewportHeight);
    glutInitWindowPosition(0, 0);
    glutSetOption(GLUT_RENDERING_CONTEXT, false ? GLUT_USE_CURRENT_CONTEXT : GLUT_CREATE_NEW_CONTEXT);
    glutCreateWindow(0);

    char m_windowsTitle[512];
    sprintf_s(m_windowsTitle, 512, "SimpleView | context %s | renderer %s | vendor %s ",
              (const char *)glGetString(GL_VERSION),
              (const char *)glGetString(GL_RENDERER),
              (const char *)glGetString(GL_VENDOR));
    glutSetWindowTitle(m_windowsTitle);

    glutDisplayFunc(my_display);
    glutReshapeFunc(my_resize);
    glutIdleFunc(my_idle);
    glutSetCursor(GLUT_CURSOR_CROSSHAIR);

    // initialize necessary OpenGL extensions
    glewInit();

    glClearColor(0.0, 0.0, 0.0, 1.0);
    glShadeModel(GL_SMOOTH);
    glViewport(0, 0, gl.viewportWidth, gl.viewportHeight);

    glFlush();
}

void releaseOpenGL()
{
    if (gl.textureID > 0)
        glDeleteTextures(1, &gl.textureID);
    if (gl.pboID > 0)
        glDeleteBuffers(1, &gl.pboID);
}
#pragma endregion OpenGL Routines

void releaseResources()
{
    releaseCUDA();
    releaseOpenGL();
}

int main(int argc, char *argv[])
{
    initializeCUDA(deviceProp);
    FreeImage_Initialise();

    initGL(argc, argv);
    prepareGlObjects("C:/Users/matej/Desktop/Source/pa2/lena.png");

    initCUDAObjects();

    // start rendering mainloop
    glutMainLoop();
    FreeImage_DeInitialise();
    atexit(releaseResources);
}
