#include "hip/hip_runtime.h"
#include <cudaDefs.h>
#include <time.h>
#include <math.h>
#include <benchmark.h>

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

constexpr unsigned int N = 1 << 20;
constexpr unsigned int MEMSIZE = N * sizeof(unsigned int);
constexpr unsigned int NO_LOOPS = 100;
constexpr unsigned int TPB = 256;
constexpr unsigned int GRID_SIZE = (N + TPB - 1) / TPB;

constexpr unsigned int NO_TEST_PHASES = 10;

void fillData(unsigned int* data, const unsigned int length)
{
	for (unsigned int i = 0; i < length; i++)
	{
		data[i] = 1;
	}
}

void printData(const unsigned int* data, const unsigned int length)
{
	if (data == 0) return;
	for (unsigned int i = 0; i < length; i++)
	{
		printf("%u ", data[i]);
	}
}


__global__ void kernel(const unsigned int* a, const unsigned int* b, const unsigned int length, unsigned int* c)
{
	const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = tid; i < length; i += stride)
		c[i] = a[i] + b[i];
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Tests 1. - single stream, async calling </summary>
////////////////////////////////////////////////////////////////////////////////////////////////////
void test1()
{
	unsigned int* a, * b, * c;
	unsigned int* da, * db, * dc;

	// paged-locked allocation
	checkCudaErrors(hipHostAlloc((void**)&a, NO_LOOPS * MEMSIZE, hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void**)&b, NO_LOOPS * MEMSIZE, hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void**)&c, NO_LOOPS * MEMSIZE, hipHostMallocDefault));

	fillData(a, NO_LOOPS * N);
	fillData(b, NO_LOOPS * N);

	// Data chunks on GPU
	checkCudaErrors(hipMalloc((void**)&da, MEMSIZE));
	checkCudaErrors(hipMalloc((void**)&db, MEMSIZE));
	checkCudaErrors(hipMalloc((void**)&dc, MEMSIZE));

	// Create stream
	hipStream_t stream1;
	checkCudaErrors(hipStreamCreate(&stream1));
	
	auto lambda = [&]()
		{
			unsigned int dataOffset = 0;
			for (int i = 0; i < NO_LOOPS; i++)
			{
				// copy a->da, b->db
				checkCudaErrors(hipMemcpyAsync(da, &a[dataOffset], MEMSIZE, hipMemcpyHostToDevice, stream1));
				checkCudaErrors(hipMemcpyAsync(db, &b[dataOffset], MEMSIZE, hipMemcpyHostToDevice, stream1));

				// run the kernel in the stream
				kernel<<<GRID_SIZE, TPB, 0, stream1>>>(da, db, N, dc);

				// copy dc->c
				checkCudaErrors(hipMemcpyAsync(&c[dataOffset], dc, MEMSIZE, hipMemcpyDeviceToHost, stream1));

				dataOffset += N;
			}
		};
	float gpuTime = GPUTIME(NO_TEST_PHASES, lambda());

	hipStreamSynchronize(stream1); // wait for stream to finish
	hipStreamDestroy(stream1);
	hipDeviceSynchronize();
	printf("\x1B[93m[GPU time] %s: %f ms\033[0m\n", __PRETTY_FUNCTION__, gpuTime);

	printData(c, 100);

	hipFree(da);
	hipFree(db);
	hipFree(dc);

	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Tests 2. - two streams - depth first approach </summary>
////////////////////////////////////////////////////////////////////////////////////////////////////
void test2()
{
	// Depth-wise data parallelism
	unsigned int* a, * b, * c;
	unsigned int* da1, * db1, * dc1;
	unsigned int* da2, * db2, * dc2;

	// paged-locked allocation
	checkCudaErrors(hipHostAlloc((void**)&a, NO_LOOPS * MEMSIZE, hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void**)&b, NO_LOOPS * MEMSIZE, hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void**)&c, NO_LOOPS * MEMSIZE, hipHostMallocDefault));

	fillData(a, NO_LOOPS * N);
	fillData(b, NO_LOOPS * N);

	// Data chunks on GPU
	checkCudaErrors(hipMalloc((void**)&da1, MEMSIZE));
	checkCudaErrors(hipMalloc((void**)&db1, MEMSIZE));
	checkCudaErrors(hipMalloc((void**)&dc1, MEMSIZE));

	checkCudaErrors(hipMalloc((void**)&da2, MEMSIZE));
	checkCudaErrors(hipMalloc((void**)&db2, MEMSIZE));
	checkCudaErrors(hipMalloc((void**)&dc2, MEMSIZE));

	// Create streams
	hipStream_t stream1;
	hipStream_t stream2;
	checkCudaErrors(hipStreamCreate(&stream1));
	checkCudaErrors(hipStreamCreate(&stream2));

	auto lambda = [&]()
		{
			unsigned int dataOffset = 0;
			for (int i = 0; i < NO_LOOPS; i+=2)
			{
				// Stream1
				//>>>>>>>>>
				checkCudaErrors(hipMemcpyAsync(da1, &a[dataOffset], MEMSIZE, hipMemcpyHostToDevice, stream1));
				checkCudaErrors(hipMemcpyAsync(db1, &b[dataOffset], MEMSIZE, hipMemcpyHostToDevice, stream1));

				kernel<<<GRID_SIZE, TPB, 0, stream1>>>(da1, db1, N, dc1);

				checkCudaErrors(hipMemcpyAsync(&c[dataOffset], dc1, MEMSIZE, hipMemcpyDeviceToHost, stream1));

				// Stream2
				//>>>>>>>>>
				dataOffset += N;

				checkCudaErrors(hipMemcpyAsync(da2, &a[dataOffset], MEMSIZE, hipMemcpyHostToDevice, stream2));
				checkCudaErrors(hipMemcpyAsync(db2, &b[dataOffset], MEMSIZE, hipMemcpyHostToDevice, stream2));

				kernel<<<GRID_SIZE, TPB, 0, stream2>>>(da2, db2, N, dc2);

				checkCudaErrors(hipMemcpyAsync(&c[dataOffset], dc2, MEMSIZE, hipMemcpyDeviceToHost, stream2));

				dataOffset += N;
			}
		};
	float gpuTime = GPUTIME(NO_TEST_PHASES, lambda());

	hipStreamSynchronize(stream1); // wait for stream to finish
	hipStreamSynchronize(stream2); // wait for stream to finish

	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);

	hipDeviceSynchronize();
	printf("\x1B[93m[GPU time] %s: %f ms\033[0m\n", __PRETTY_FUNCTION__, gpuTime);

	printData(c, 100);

	hipFree(da1);
	hipFree(db1);
	hipFree(dc1);

	hipFree(da2);
	hipFree(db2);
	hipFree(dc2);

	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Tests 3. - two streams - breadth first approach</summary>
////////////////////////////////////////////////////////////////////////////////////////////////////
void test3()
{
	// Breadth first data parallelism
	unsigned int* a, * b, * c;
	unsigned int* da1, * db1, * dc1;
	unsigned int* da2, * db2, * dc2;

	// paged-locked allocation
	checkCudaErrors(hipHostAlloc((void**)&a, NO_LOOPS * MEMSIZE, hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void**)&b, NO_LOOPS * MEMSIZE, hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void**)&c, NO_LOOPS * MEMSIZE, hipHostMallocDefault));

	fillData(a, NO_LOOPS * N);
	fillData(b, NO_LOOPS * N);

	// Data chunks on GPU
	checkCudaErrors(hipMalloc((void**)&da1, MEMSIZE));
	checkCudaErrors(hipMalloc((void**)&db1, MEMSIZE));
	checkCudaErrors(hipMalloc((void**)&dc1, MEMSIZE));

	checkCudaErrors(hipMalloc((void**)&da2, MEMSIZE));
	checkCudaErrors(hipMalloc((void**)&db2, MEMSIZE));
	checkCudaErrors(hipMalloc((void**)&dc2, MEMSIZE));

	// Create streams
	hipStream_t stream1;
	hipStream_t stream2;
	checkCudaErrors(hipStreamCreate(&stream1));
	checkCudaErrors(hipStreamCreate(&stream2));

	auto lambda = [&]()
		{
			unsigned int dataOffset1 = 0;
			unsigned int dataOffset2 = N;
			for (int i = 0; i < NO_LOOPS; i += 2)
			{
				// a -> da
				checkCudaErrors(hipMemcpyAsync(da1, &a[dataOffset1], MEMSIZE, hipMemcpyHostToDevice, stream1));
				checkCudaErrors(hipMemcpyAsync(da2, &a[dataOffset2], MEMSIZE, hipMemcpyHostToDevice, stream2));

				// b -> db
				checkCudaErrors(hipMemcpyAsync(db1, &b[dataOffset1], MEMSIZE, hipMemcpyHostToDevice, stream1));
				checkCudaErrors(hipMemcpyAsync(db2, &b[dataOffset2], MEMSIZE, hipMemcpyHostToDevice, stream2));

				// enqueue kernel for stream1 and stream2 (async by default)
				kernel<<<GRID_SIZE, TPB, 0, stream1>>>(da1, db1, N, dc1);
				kernel<<<GRID_SIZE, TPB, 0, stream2>>>(da2, db2, N, dc2);

				// dc -> c
				checkCudaErrors(hipMemcpyAsync(&c[dataOffset1], dc1, MEMSIZE, hipMemcpyDeviceToHost, stream1));
				checkCudaErrors(hipMemcpyAsync(&c[dataOffset2], dc2, MEMSIZE, hipMemcpyDeviceToHost, stream2));

				dataOffset1 += N;
				dataOffset2 += N;
			}
		};
	float gpuTime = GPUTIME(NO_TEST_PHASES, lambda());

	hipStreamSynchronize(stream1); // wait for stream to finish
	hipStreamSynchronize(stream2); // wait for stream to finish

	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);

	hipDeviceSynchronize();
	printf("\x1B[93m[GPU time] %s: %f ms\033[0m\n", __PRETTY_FUNCTION__, gpuTime);

	printData(c, 100);

	hipFree(da1);
	hipFree(db1);
	hipFree(dc1);

	hipFree(da2);
	hipFree(db2);
	hipFree(dc2);

	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);
}


int main(int argc, char* argv[])
{
	initializeCUDA(deviceProp);

	test1();
	test2();
	test3();

	return 0;
}
