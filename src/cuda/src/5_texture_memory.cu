#include "hip/hip_runtime.h"
#include <cudaDefs.h>
#include <hip/hip_vector_types.h>  // normalize method
#include <imageManager.h>
#include <imageUtils.cuh>
#include <benchmark.h>

#define TPB_1D 8  // ThreadsPerBlock in one dimension
#define TPB_2D TPB_1D * TPB_1D  // ThreadsPerBlock = TPB_1D*TPB_1D (2D block)

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

using namespace gpubenchmark;
using DT = float;


__host__ TextureInfo createTextureObjectFrom2DArray(const ImageInfo<DT>& ii)
{
	TextureInfo ti;

	// Size info
	ti.size = { ii.width, ii.height, 1 };

	//Texture Data settings
	ti.texChannelDesc = hipCreateChannelDesc<DT>();  // hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);
	checkCudaErrors(hipMallocArray(
		&ti.texArrayData, &ti.texChannelDesc, ii.width, ii.height
	)); // allocate hipArray
	checkCudaErrors(hipMemcpyToArray(
		ti.texArrayData, 0, 0, ii.dPtr, ii.pitch * ii.height, hipMemcpyDeviceToDevice
	)); // dPtr is already on device

	// Specify texture resource
	ti.resDesc.resType = hipResourceTypeArray; // hipArray
	ti.resDesc.res.array.array = ti.texArrayData; // hipArray

	// Specify texture object parameters
	ti.texDesc.addressMode[0] = hipAddressModeClamp; // clamp to x-border
	ti.texDesc.addressMode[1] = hipAddressModeClamp; // clamp to y-border
	ti.texDesc.filterMode = hipFilterModePoint; // matrix-like access
	ti.texDesc.readMode = hipReadModeElementType; // matrix-like access
	ti.texDesc.normalizedCoords = false; // access by int coordinates [(0,h-1),(0,w-1)] or by float [(0,1),(0,1)]

	// Create texture object
	checkCudaErrors(hipCreateTextureObject(
		&ti.texObj, &ti.resDesc, &ti.texDesc, nullptr
	)); // nullptr or change channel order (BGR/RGB)

	return ti;
}

__global__ void texKernel(
	const hipTextureObject_t srcTex,
	const unsigned int srcWidth, 
	const unsigned int srcHeight, 
	float* dst)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < srcWidth && y < srcHeight)
	{
		dst[y * srcWidth + x] = tex2D<float>(srcTex, x, y);
	}
}

int main(int argc, char* argv[])
{
	initializeCUDA(deviceProp);
	FreeImage_Initialise(); // Initialize the FreeImage library

	// STEP 1 - load raw image data, HOST->DEVICE, with/without pitch
	ImageInfo<DT> src;
	// false - without pitch,
	prepareData<false>("c:/Users/matej/Desktop/Source/pa2/textures/terrain10x10.tif", src);

	// STEP 2 - create texture from the raw data
	TextureInfo tiSrc = createTextureObjectFrom2DArray(src);

	// STEP 3 - DO SOMETHING WITH THE TEXTURE
	dim3 block = {TPB_1D, TPB_1D, 1};
	dim3 grid{ 
		(src.width + TPB_1D - 1) / TPB_1D, 
		(src.height + TPB_1D - 1) / TPB_1D,
		1 
	};
	float* dst = nullptr;
	hipMalloc((void**)&dst, src.width * src.height * sizeof(float));
	float gpuTime = GPUTIME(1, 
		texKernel<<<grid, block>>>(tiSrc.texObj, src.width, src.height, dst)
	);
	printf("\x1B[93m[GPU time] %s: %f ms\033[0m\n", "getBest", gpuTime);
	checkDeviceMatrix<float>(dst, src.width * sizeof(float), src.height, src.width, "%6.1f ", "dst");

	// STEP 4 - release unused data
	if (tiSrc.texObj)
		// Check that memory freed without errors.
		checkCudaErrors(hipDestroyTextureObject(tiSrc.texObj));
	if (tiSrc.texArrayData)
		checkCudaErrors(hipFreeArray(tiSrc.texArrayData));
	if (src.dPtr) hipFree(src.dPtr);
	if (dst) hipFree(dst);

	hipDeviceSynchronize(); // Wait for the GPU launched work to complete
	error = hipGetLastError();

	FreeImage_DeInitialise();
}
